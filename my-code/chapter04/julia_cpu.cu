#include "../../example_code/common/cpu_bitmap.h"
#include "../../example_code/common/book.h"
#include <chrono>
#include <iostream>

#define DIM 3000

struct hipComplex {
    float r;
    float i;
    hipComplex(float a, float b): r(a), i(b) {}
    float magnitude2(void) {return r*r + i*i;}
    hipComplex operator*(const hipComplex &in)
    {
        return hipComplex(r*in.r - i*in.i, i*in.r + r * in.i);
    }
    hipComplex operator+(const hipComplex &in)
    {
        return hipComplex(r + in.r, i + in.i);
    }

};

int julia(int x, int y)
{
    const float scale = 1.5;
    float jx =scale * (float)(DIM/2-x)/(DIM/2);
    float jy =scale * (float)(DIM/2-y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    for(int i =0; i < 200; i++) {
        a = a*a +c;
        if (a.magnitude2() > 1000)
            return 0;
    }
    return 1;
}

void kernel(unsigned char* ptr)
{
    for(int y = 0; y < DIM; y++) {
        for (int x = 0; x < DIM; x++) {
            int offset = x + y * DIM;
            int juliaValue = julia(x, y);

            ptr[offset*4 + 0] = 255 * juliaValue;
            ptr[offset*4 + 1] = 0;
            ptr[offset*4 + 2] = 0;
            ptr[offset*4 + 3] = 255;
                
        }
    }

}


int main(void)
{
    CPUBitmap bitmap(DIM, DIM);
    unsigned char *ptr = bitmap.get_ptr();

    auto start = std::chrono::high_resolution_clock::now();
    kernel(ptr);
    auto stop = std::chrono::high_resolution_clock::now();
    std::cout << "Run time: " << ((float)std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count())/1000000.0 << std::endl;
    
    bitmap.display_and_exit(); 
    
}