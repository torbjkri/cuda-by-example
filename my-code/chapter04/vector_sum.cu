#include "hip/hip_runtime.h"
#include "../../example_code/common/book.h"


#define N 10

__global__ void add(int* a, int *b, int *c)
{
    int tid = blockIdx.x; //task ID? this is CPU code so we only got 1, CPU 0
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main (void)
{
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a, sizeof(int)*N));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, sizeof(int)*N));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)*N));

    // fill a and b with data on CPU
    for(int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i*i;
    }

    HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));


    add<<<N,1>>>(dev_a,dev_b,dev_c);

    HANDLE_ERROR(hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));
    for(int i = 0; i<N;i++) {
        printf("%d + %d = %d\n",a[i],b[i],c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}